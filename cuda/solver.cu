#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, Dirk Pleiter, KTH
 *
 * This source code is in parts based on code from Jiri Kraus (NVIDIA) and
 * Andreas Herten (Forschungszentrum Juelich)
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND Any
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR Any DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON Any THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN Any WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
 #ifndef NX
 #define NX 128
 #endif
 #ifndef NY
 #define NY 128
 #endif
 #define NMAX 2000000
 #define EPS 1e-5
 #define N_SKIP 100
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <math.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include "helper.h"
/**
* @brief https://leimao.github.io/blog/CUDA-Shared-Memory-Capacity/
* 
*/
#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, char const* const func, char const* const file,
		int const line)
{
	if (err != hipSuccess)
	{
		std::cerr << "CUDA Runtime Error at: " << file << ":" << line
				<< std::endl;
		std::cerr << hipGetErrorString(err) << " " << func << std::endl;
		std::exit(EXIT_FAILURE);
	}
}
#define CHECK_LAST_CUDA_ERROR() checkLast(__FILE__, __LINE__)
void checkLast(char const* const file, int const line)
{
	hipError_t err{hipGetLastError()};
	if (err != hipSuccess)
	{
		std::cerr << "CUDA Runtime Error at: " << file << ":" << line
				<< std::endl;
		std::cerr << hipGetErrorString(err) << std::endl;
		std::exit(EXIT_FAILURE);
	}
}

__global__
void jacobi_stride(double *v, double *vp, double *f, int X, int Y){
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	// int x = index % X;
	// int y = index / X;
	for(size_t i = index; i < X*Y; i += stride){
		int x = i % X;
		int y = i / X;
		if(x == 0 || y == 0 || y >= Y-1 || x >= X-1) continue;
		vp[y*X+x] = -0.25 * (f[y*X+x] - (v[X*y+x+1] + v[X*y+x-1] + v[X*(y+1) + x  ] + v[X*(y-1) + x  ]));
	}
}

__global__
void jacobi(double *v, double *vp, double *f, int X, int Y){
	int tid = threadIdx.x + blockIdx.x* blockDim.x;
	// if(tid >= X*Y) return;
	int x = tid % X;
	int y = tid / X;
	if(x == 0 || y == 0 || y >= Y-1 || x >= X-1) return;
    vp[y*X+x] = -0.25 * (f[y*X+x] - (v[X*y+x+1] + v[X*y+x-1] + v[X*(y+1) + x  ] + v[X*(y-1) + x  ]));
    
    // __syncthreads();
    // if(x==1)
    //     vp[X*y + (X-1)] = vp[X*y + 1     ];
    // if(y==1)
    //     vp[X*(Y-1) + x] = vp[X*1      + x];
    // if(x==X-2)
    //     vp[X*y + 0]      = vp[X*y + (X-2)];
    // if(y==Y-2)
    //     vp[X*0      + x] = vp[X*(Y-2) + x];
    // printf("(%d, %d)=%f\n", x,y,vp[y*X+x]);
	// grid_new[y*X+x]=(grid_old[(y-1)*X+x]+grid_old[(y+1)*X+x]+grid_old[y*X+x-1]+grid_old[y*X+x+1]);
}

__global__
void jacobi_bounds(double *v, int X, int Y){
	int tid = threadIdx.x + blockIdx.x* blockDim.x;
	int ix = tid % X;
	int iy = tid / X;
    
    if(iy == 0 && ix > 0 && ix < X -1)
        v[X*0      + ix] = v[X*(Y-2) + ix];
    if(iy == Y-1&& ix > 0 && ix < X -1)
        v[X*(Y-1) + ix] = v[X*1      + ix];
    if(ix == 0&& iy > 0&& iy < Y -1)
        v[X*iy + 0]      = v[X*iy + (X-2)];
    if(ix == X-1&& iy > 0&& iy < Y -1)
        v[X*iy + (X-1)] = v[X*iy + 1     ];
}
__global__
void calcDs(double *v, double *vp, double *ds, int X, int Y){
	int tid = threadIdx.x + blockIdx.x* blockDim.x;
	// if(tid >= X*Y) return;
	int x = tid % X;
	int y = tid / X;
	if(x == 0 || y == 0 || y >= Y-1 || x >= X-1) return;
    // vp[y*X+x] = -0.25 * (f[y*X+x] - (v[X*y+x+1] + v[X*y+x-1] + v[X*(y+1) + x  ] + v[X*(y-1) + x  ]));
    ds[y*X+x] = fabs(vp[y*X+x] - v[y*X+x]);
    
    // printf("(%d, %d)=%f\n", x,y,vp[y*X+x]);
	// grid_new[y*X+x]=(grid_old[(y-1)*X+x]+grid_old[(y+1)*X+x]+grid_old[y*X+x-1]+grid_old[y*X+x+1]);
}
// __device__ double w_answer;

double calcEps(double *v, double *vp, size_t nx, size_t ny){
    double e = 0.0;
    double w = 0.0;
    for( int ix = 1; ix < (nx-1); ix++ )
    {
        for (int iy = 1; iy < (ny-1); iy++)
        {
            double d;

            d = fabs(vp[nx*iy+ix] - v[nx*iy+ix]);
            e = (d > e) ? d : e;
            w += fabs(v[nx*iy+ix]);
        }
    }

    for (int ix = 1; ix < (nx-1); ix++)
    {
        w += fabs(v[nx*0+ix]) + fabs(v[nx*(ny-1)+ix]);
    }

    for (int iy = 1; iy < (ny-1); iy++)
    {
        w += fabs(v[nx*iy+0]) + fabs(v[nx*iy+(nx-1)]);
    }
    // printf("Sum %f \n", w);
    w /= (nx * ny);
    e /= w;
    return e;
}

template<typename T>
struct absolute_value
{
  __host__ __device__ T operator()(const T &x) const
  {
    // return x < T(0) ? -x : x;
    return fabs(x);
  }
};

int solver(double *v, double *f, int nx, int ny, double eps, int nmax)
{
    int n = 0;
    double e = 2. * eps;
    double *vp, *ds;
    size_t bytes = nx * ny * sizeof(double);
	hipMallocManaged(&vp, bytes);
    CHECK_CUDA_ERROR(hipMalloc(&ds, bytes));
    thrust::device_ptr<double> ds_thrust(ds);
    thrust::device_ptr<double> v_thrust(v);

    int deviceId; // TODO repetition. see main()
	int numberOfSMs;
	hipGetDevice(&deviceId);
	hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
    CHECK_CUDA_ERROR(hipMemPrefetchAsync(v, bytes, deviceId));
    CHECK_CUDA_ERROR(hipMemPrefetchAsync(f, bytes, deviceId));
    CHECK_CUDA_ERROR(hipMemPrefetchAsync(vp, bytes, deviceId));
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    int threadsPerBlock = 1024;
    int numberOfBlocks = ((nx*ny) + threadsPerBlock - 1) / threadsPerBlock; 
    // double e = eps*2;
    double n_count = 0;
    while (( e > eps) && (n < nmax))
    {
        // printf("e: %f, eps: %f \n", e,eps);
        jacobi<<<numberOfBlocks,threadsPerBlock>>>(v,vp,f,nx,ny); //TODO change size
        CHECK_LAST_CUDA_ERROR();
        jacobi_bounds<<<numberOfBlocks,threadsPerBlock>>>(vp,nx,ny); //TODO change size
        CHECK_LAST_CUDA_ERROR();
        
        // CHECK_CUDA_ERROR(hipDeviceSynchronize());
        // e = calcEps(v,vp,nx,ny);

        jacobi<<<numberOfBlocks,threadsPerBlock>>>(vp,v,f,nx,ny); //TODO change size
        CHECK_LAST_CUDA_ERROR();
        jacobi_bounds<<<numberOfBlocks,threadsPerBlock>>>(v,nx,ny); //TODO change size
        CHECK_LAST_CUDA_ERROR();
        
        if(n_count >= N_SKIP){
            calcDs<<<numberOfBlocks,threadsPerBlock>>>(v,vp,ds,nx,ny);
            e = thrust::transform_reduce(&ds_thrust[0], &ds_thrust[nx * ny],absolute_value<double>(), (double) 0.0, thrust::maximum<double>());
            double w = thrust::transform_reduce(&v_thrust[0], &v_thrust[nx * ny],absolute_value<double>(), (double) 0.0, thrust::plus<double>());
            // // // double sum = thrust::reduce(&v[0], &v[nx * ny],(double) 0.0, thrust::plus<double>());
            // // // printf("CUDA_Sum %f \n", w);
            // // printf("CUDA_Sum %f \n", e);
            w /= (nx * ny);
            e /= w;
            // printf("CUDA_Sum %f \n", e);
            n_count =0;
            // CHECK_CUDA_ERROR(hipDeviceSynchronize());
            // CHECK_CUDA_ERROR(hipMemPrefetchAsync(vp, bytes, -1));
            
            // e = calcEps(v,vp,nx,ny);
            // printf("Sum %f \n", e);
            // CHECK_CUDA_ERROR(hipMemPrefetchAsync(vp, bytes, deviceId));
        }
        n_count ++;
        n++;
    }
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    e = calcEps(v,vp,nx,ny);

	hipFree(vp);

    if (e < eps)
        printf("Converged after %d iterations (nx=%d, ny=%d, e=%.2e)\n", n, nx, ny, e);
    else
        printf("ERROR: Failed to converge\n");

    return (e < eps ? 0 : 1);
}


// TODO use CUBLAS "gesvdj" as reference
int main()
{
    // printf("OMP-Threads: %d\n", omp_get_max_threads());
    double *v;
    double *f;
    size_t bytes = NX * NY * sizeof(double);
	hipMallocManaged(&v, bytes);
	hipMallocManaged(&f, bytes);

    // Allocate memory
    // v = (double *) malloc(NX * NY * sizeof(double));
    // f = (double *) malloc(NX * NY * sizeof(double));

    // Initialise input
    for (int iy = 0; iy < NY; iy++)
        for (int ix = 0; ix < NX; ix++)
        {
            v[NX*iy+ix] = 0.0;

            const double x = 2.0 * ix / (NX - 1.0) - 1.0;
            const double y = 2.0 * iy / (NY - 1.0) - 1.0;
            f[NX*iy+ix] = sin(x + y);
        }

	int deviceId;
	int numberOfSMs;
	hipGetDevice(&deviceId);
	hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
    CHECK_CUDA_ERROR(hipMemPrefetchAsync(v, bytes, deviceId));
    CHECK_CUDA_ERROR(hipMemPrefetchAsync(f, bytes, deviceId));
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    hipEvent_t start, stop;     		// using cuda events to measure time
	float elapsed_time_ms; 
    hipEventCreate(&start);     		// events to measure start time
	hipEventCreate(&stop);  	


    // // Call solver
    printf("Solver:\n");
	hipEventRecord(start, 0);
    solver(v, f, NX, NY, EPS, NMAX);
    CHECK_LAST_CUDA_ERROR();
	hipEventRecord(stop, 0);     	// instrument code to measue end time
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time_ms, start, stop );
    printf("Took %f sec\n", elapsed_time_ms/1000);
    compareToFile(v,NX,NY,NMAX,EPS);


    
    // Clean-up
    hipFree(v);
    hipFree(f);

    return 0;
}
