#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, Dirk Pleiter, KTH
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "helperMPI.h"
#include <string.h>
// #include <omp.h>
#include <mpi.h>
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <mpi-ext.h>
#include <nvml.h>
// #include <nvidia/gdk/nvml.h>
#ifndef NX
#define NX 1024
#endif
#ifndef NY
#define NY 1024
#endif
#define NMAX 2000000
#define EPS 1e-5
#define N_SKIP 100

int solver_mpi(double *,double *, double *, int, int, double, int,MPI_Comm,int,int);

#define IF_MASTER if(rank==0)

int main(int argc, char** argv)
{
    MPI_Init(&argc, &argv);
    if (MPIX_Query_cuda_support()) {
        printf("This MPI library has CUDA-aware support.\n");
    } else {
        printf("This MPI library does not have CUDA-aware support.\n");
        exit(EXIT_FAILURE);
    }
    int world_size;
    MPI_Comm_size(MPI_COMM_WORLD, &world_size);
    int rank;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    // IF_MASTER printf("OMP-Threads: %d\n", omp_get_max_threads());
    IF_MASTER printf("MPI-tasks:  %d\n", world_size);



     // Ask MPI to decompose our processes in a 2D cartesian grid for us
    int dims[2] = {0, 0};
    MPI_Dims_create(world_size, 2, dims);
    IF_MASTER printf("DIMS (%d, %d)\n",dims[0],dims[1]);
    // Make both dimensions periodic
    int periods[2] = {true, true};
    // Let MPI assign arbitrary ranks if it deems it necessary
    int reorder = true;
    // Create a communicator given the 2D torus topology.
    MPI_Comm new_communicator;
    MPI_Cart_create(MPI_COMM_WORLD, 2, dims, periods, reorder, &new_communicator);    
    // My rank in the new communicator
    int my_rank;
    MPI_Comm_rank(new_communicator, &my_rank);
    // Get my coordinates in the new communicator
    int my_coords[2];
    MPI_Cart_coords(new_communicator, my_rank, 2, my_coords);


    int mpiSizeX = dims[0];
    int mpiSizeY = dims[1];
    size_t NX_true = NX - 2;
    size_t NY_true = NY - 2;
    size_t NXL = NX_true/mpiSizeX;
    size_t NYL = NY_true/mpiSizeY;
    // printf("NYL: %d\n",NYL);
    size_t NXLRest = NX_true%mpiSizeX;
    size_t NYLRest = NY_true%mpiSizeY;
    if(my_coords[0]<NXLRest)NXL++;
    if(my_coords[1]<NYLRest)NYL++;
    size_t NXLStart = (NX_true/mpiSizeX) * my_coords[0];
    size_t NYLStart = (NY_true/mpiSizeY) * my_coords[1];
    if(my_coords[0]<NXLRest)NXLStart+=my_coords[0];
    else NXLStart+=NXLRest;
    if(my_coords[1]<NYLRest)NYLStart+=my_coords[1];
    else NYLStart+=NYLRest;

    // printf("NXLStart=%d \n",NXLStart);


    double *v,*v_tmp;
    double *f;

    // Allocate memory
    size_t bytes = (NXL+2) * (NYL+2) * sizeof(double);
	// hipMallocManaged(&vp, bytes);
	// hipMallocManaged(&v, bytes);
	// hipMallocManaged(&v_tmp, bytes);
	// hipMallocManaged(&f, bytes);
    // CHECK_CUDA_ERROR(hipMalloc(&ds, bytes));
    v = (double *) malloc((NXL+2) * (NYL+2) * sizeof(double));
    v_tmp = (double *) malloc((NXL+2) * (NYL+2) * sizeof(double));
    f = (double *) malloc((NXL+2) * (NYL+2) * sizeof(double));
	// v = aligned_alloc(64, (NXL+2) * (NYL+2) * sizeof(double));
	// v_tmp = aligned_alloc(64, (NXL+2) * (NYL+2) * sizeof(double));
	// f = aligned_alloc(64, (NXL+2) * (NYL+2) * sizeof(double));

    // if(rank == 0)
    // for (int iy = 0; iy < NY; iy++){
    //     for (int ix = 0; ix < NX; ix++)
    //     {
    //         const double x = 2.0 * ix / (NX - 1.0) - 1.0;
    //         const double y = 2.0 * iy / (NY - 1.0) - 1.0;
    //         double res = sin(x + y);
    //         printf("f[%d,%d]=%f, ",iy,ix,res);
    //     }
    //     printf("\n");
    // }


    // Initialise input
    for (int iy = 0; iy < (NYL+2); iy++)
        for (int ix = 0; ix < (NXL+2); ix++)
        {
            v[(NXL+2)*iy+ix] = 0.0;

            const double x = 2.0 * (ix+NXLStart) / (NX - 1.0) - 1.0;
            const double y = 2.0 * (iy+NYLStart) / (NY - 1.0) - 1.0;
            f[(NXL+2)*iy+ix] = sin(x + y);
            // printf("f[%d,%d]=%f\n",NYLStart + iy, NXLStart + ix,sin(x + y));
        }
    // Call solver
    // return 0;

    memset(v,0.0,(NXL+2) * (NYL+2)* sizeof(double));
    //     for (size_t i = 0; i < world_size; i++)
    // {
    //     MPI_Barrier(new_communicator);
    //     IF_MASTER printf("\n");
    //     MPI_Barrier(new_communicator);
    //     if(my_rank == i)printArr(f,NXL+2,NYL+2);
    // }
    IF_MASTER printf("Solver:\n");
    double start = getTime();
    solver_mpi(v,v_tmp, f, NX, NY, EPS, NMAX,new_communicator,NXL,NYL);
    double end = getTime();
    IF_MASTER printf("Took %f sec\n", end-start);
    
    // memset(v,0.0,(NXL+2) * (NYL+2)* sizeof(double));
    // memset(v_tmp,0.0,(NXL+2) * (NYL+2)* sizeof(double));
    // IF_MASTER printf("Solver AVX:\n");
    // start = getTime();
    // // solver_transposed_mpi_avx(v,v_tmp, f, NX, NY, EPS, NMAX,new_communicator,NXL,NYL);
    // end = getTime();
    // IF_MASTER printf("Took %f sec\n", end-start);
    // compareToFile(v,NX,NY,NMAX,EPS);
    
    // IF_MASTER printf("Writing to file...\n");
    writeGnuPlotFile(v,NX,NY,NXLStart,NYLStart,NXL+2,NYL+2,rank,world_size,new_communicator);
    // MPI_Barrier(new_communicator);
    // IF_MASTER printf("Writing complete\n");
    // for (size_t i = 0; i < world_size; i++)
    // {
    //     MPI_Barrier(new_communicator);
    //     // IF_MASTER printf("\n");
    //     MPI_Barrier(new_communicator);
    //     if(my_rank == i)printArr(v,NXL+2,NYL+2);
    // }
    hipFree(v);
    hipFree(v_tmp);
    hipFree(f);
    MPI_Finalize();
    return 0;
}


#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, char const* const func, char const* const file,
		int const line)
{
	if (err != hipSuccess)
	{
		std::cerr << "CUDA Runtime Error at: " << file << ":" << line
				<< std::endl;
		std::cerr << hipGetErrorString(err) << " " << func << std::endl;
		std::exit(EXIT_FAILURE);
	}
}
#define CHECK_LAST_CUDA_ERROR() checkLast(__FILE__, __LINE__)
void checkLast(char const* const file, int const line)
{
	hipError_t err{hipGetLastError()};
	if (err != hipSuccess)
	{
		std::cerr << "CUDA Runtime Error at: " << file << ":" << line
				<< std::endl;
		std::cerr << hipGetErrorString(err) << std::endl;
		std::exit(EXIT_FAILURE);
	}
}


__global__
void jacobi(double *v, double *vp, double *f, int X, int Y){
	int tid = threadIdx.x + blockIdx.x* blockDim.x;
	// if(tid >= X*Y) return;
	int x = tid % X;
	int y = tid / X;
	if(x == 0 || y == 0 || y >= Y-1 || x >= X-1) return;
    vp[y*X+x] = -0.25 * (f[y*X+x] - (v[X*y+x+1] + v[X*y+x-1] + v[X*(y+1) + x  ] + v[X*(y-1) + x  ]));
    
    // __syncthreads();
    // if(x==1)
    //     vp[X*y + (X-1)] = vp[X*y + 1     ];
    // if(y==1)
    //     vp[X*(Y-1) + x] = vp[X*1      + x];
    // if(x==X-2)
    //     vp[X*y + 0]      = vp[X*y + (X-2)];
    // if(y==Y-2)
    //     vp[X*0      + x] = vp[X*(Y-2) + x];
    // printf("(%d, %d)=%f\n", x,y,vp[y*X+x]);
	// grid_new[y*X+x]=(grid_old[(y-1)*X+x]+grid_old[(y+1)*X+x]+grid_old[y*X+x-1]+grid_old[y*X+x+1]);
}
__global__
void jacobi_fillRightLeftBounds(double *v, double *v_left, double *v_right, int X, int Y){
	int tid = threadIdx.x + blockIdx.x* blockDim.x;
	// if(tid >= X*Y) return;
	int x = tid % X;
	int y = tid / X;
	if(! ( y > 0 && y < Y - 1) )return;
	if( x == 1 ){
        v_left[y-1] = v[y*X+x];
    } else if( x == X - 2){
        v_right[y-1] = v[y*X+x];
    }
}
__global__
void jacobi_rewriteRightLeftBounds(double *v, double *v_left, double *v_right, int X, int Y){
	int tid = threadIdx.x + blockIdx.x* blockDim.x;
	// if(tid >= X*Y) return;
	int x = tid % X;
	int y = tid / X;
	if(! ( y > 0 && y < Y - 1) )return;
	if( x == 0 ){
        v[y*X+x] = v_left[y-1];
    } else if( x == X - 1){
        v[y*X+x] = v_right[y-1];
    }
    
}
__global__
void jacobi_bounds(double *v, int X, int Y){
	int tid = threadIdx.x + blockIdx.x* blockDim.x;
	int ix = tid % X;
	int iy = tid / X;
    
    if(iy == 0 && ix > 0 && ix < X -1)
        v[X*0      + ix] = v[X*(Y-2) + ix];
    if(iy == Y-1&& ix > 0 && ix < X -1)
        v[X*(Y-1) + ix] = v[X*1      + ix];
    if(ix == 0&& iy > 0&& iy < Y -1)
        v[X*iy + 0]      = v[X*iy + (X-2)];
    if(ix == X-1&& iy > 0&& iy < Y -1)
        v[X*iy + (X-1)] = v[X*iy + 1     ];
}
__global__
void calcDs(double *v, double *vp, double *ds, int X, int Y){
	int tid = threadIdx.x + blockIdx.x* blockDim.x;
	// if(tid >= X*Y) return;
	int x = tid % X;
	int y = tid / X;
	if(x == 0 || y == 0 || y >= Y-1 || x >= X-1) return;
    // vp[y*X+x] = -0.25 * (f[y*X+x] - (v[X*y+x+1] + v[X*y+x-1] + v[X*(y+1) + x  ] + v[X*(y-1) + x  ]));
    ds[y*X+x] = fabs(vp[y*X+x] - v[y*X+x]);
    
    // printf("(%d, %d)=%f\n", x,y,vp[y*X+x]);
	// grid_new[y*X+x]=(grid_old[(y-1)*X+x]+grid_old[(y+1)*X+x]+grid_old[y*X+x-1]+grid_old[y*X+x+1]);
}

template<typename T>
struct absolute_value
{
  __host__ __device__ T operator()(const T &x) const
  {
    // return x < T(0) ? -x : x;
    return fabs(x);
  }
};

int solver_mpi(double *v_CPU,double *vp_CPU, double *f_CPU, int nx, int ny, double eps, int nmax, MPI_Comm comm, int NXL, int NYL )
{
    int rank;
    MPI_Comm_rank(comm, &rank);


    int devices;
    hipGetDeviceCount(&devices);
    hipSetDevice( rank % devices);      
    int deviceId; 
	hipGetDevice(&deviceId);
    printf("CUDA device: %d of %d\n",deviceId,devices);

    double *ds;
    size_t bytes = (NXL+2) * (NYL+2) * sizeof(double);
	// hipMallocManaged(&vp, bytes);
    CHECK_CUDA_ERROR(hipMalloc(&ds, bytes));
    double  *v,*vp,*f;
    CHECK_CUDA_ERROR(hipMalloc(&v, bytes));
    CHECK_CUDA_ERROR(hipMalloc(&vp, bytes));
    CHECK_CUDA_ERROR(hipMalloc(&f, bytes));
    CHECK_CUDA_ERROR(hipMemcpy(v, v_CPU, bytes, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(vp, vp_CPU, bytes, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(f, f_CPU, bytes, hipMemcpyHostToDevice));
    double  *v_right,*v_right_recv;
    double  *v_left,*v_left_recv;
    CHECK_CUDA_ERROR(hipMalloc(&v_right, (NYL) * sizeof(double)));
    CHECK_CUDA_ERROR(hipMalloc(&v_left, (NYL) * sizeof(double)));
    CHECK_CUDA_ERROR(hipMalloc(&v_right_recv, (NYL) * sizeof(double)));
    CHECK_CUDA_ERROR(hipMalloc(&v_left_recv, (NYL) * sizeof(double)));

    thrust::device_ptr<double> ds_thrust(ds);
    thrust::device_ptr<double> v_thrust(v);
    
    int numberOfSMs;
	hipGetDevice(&deviceId);
    // hipSetDevice( rank );  
    // hipSetDevice( 0 );                                                                 
    // hipError_t ret1 = hipDeviceEnablePeerAccess ( 1, 0);                                          
    // hipSetDevice( 1 );                                                                 
    // hipError_t ret2 = hipDeviceEnablePeerAccess (0, 0);
	hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
    // CHECK_CUDA_ERROR(hipMemPrefetchAsync(v, bytes, deviceId));
    // CHECK_CUDA_ERROR(hipMemPrefetchAsync(f, bytes, deviceId));
    // CHECK_CUDA_ERROR(hipMemPrefetchAsync(vp, bytes, deviceId));
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    int threadsPerBlock = 1024;
    int numberOfBlocks = ((nx*ny) + threadsPerBlock - 1) / threadsPerBlock; 



    // hipSetDevice( rank );  
	// hipGetDevice(&deviceId);
    // Get my coordinates in the new communicator
    int dims[2], periods[2], coords[2];
    MPI_Cart_get(comm, 2, dims, periods, coords);

    int left, right, top, bot;
    MPI_Cart_shift(comm, 0, 1, &left, &right);
    MPI_Cart_shift(comm, 1, 1, &top, &bot);
    int isLeftBorderTile = coords[0]==0;
    int isRightBorderTile = coords[0]==dims[0]-1;
    int isTopBorderTile = coords[1]==0;
    int isBottomBorderTile = coords[1]==dims[1]-1;

    int n = 0;
    double e = 2. * eps;
    
    MPI_Datatype vectorTypeVertical;
    MPI_Type_vector(NYL, 1, NXL + 2, MPI_DOUBLE, &vectorTypeVertical);
    MPI_Datatype contiguousTypeHorizontal;
    MPI_Type_contiguous(NXL, MPI_DOUBLE, &contiguousTypeHorizontal);
    MPI_Datatype contiguousTypeVertical;
    MPI_Type_contiguous(NYL, MPI_DOUBLE, &contiguousTypeVertical);
    MPI_Type_commit(&vectorTypeVertical);
    MPI_Type_commit(&contiguousTypeHorizontal);
    MPI_Type_commit(&contiguousTypeVertical);
    int n_count = 0;
    while ((e > eps) && (n < nmax))
    {
        jacobi<<<numberOfBlocks,threadsPerBlock>>>(v,vp,f,NXL+2,NYL+2); //TODO change size
        CHECK_LAST_CUDA_ERROR();
        jacobi_fillRightLeftBounds<<<numberOfBlocks,threadsPerBlock>>>(v,v_left,v_right,NXL+2,NYL+2);
        CHECK_LAST_CUDA_ERROR();
        // jacobi_bounds<<<numberOfBlocks,threadsPerBlock>>>(vp,NXL,NYL); //TODO change size
        // CHECK_LAST_CUDA_ERROR();
        
        // jacobi<<<numberOfBlocks,threadsPerBlock>>>(vp,v,f,NXL,NYL); //TODO change size
        // CHECK_LAST_CUDA_ERROR();
        // jacobi_bounds<<<numberOfBlocks,threadsPerBlock>>>(v,NXL,NYL); //TODO change size
        // CHECK_LAST_CUDA_ERROR();
        
        // if(n_count >= N_SKIP){
            // // // double sum = thrust::reduce(&v[0], &v[nx * ny],(double) 0.0, thrust::plus<double>());
            // // // printf("CUDA_Sum %f \n", w);
            // // printf("CUDA_Sum %f \n", e);
            // w /= (nx * ny);
            // e /= w;
            // printf("CUDA_Sum %f \n", e);
            // n_count =0;
            // CHECK_CUDA_ERROR(hipDeviceSynchronize());
            // CHECK_CUDA_ERROR(hipMemPrefetchAsync(vp, bytes, -1));
            
            // e = calcEps(v,vp,nx,ny);
            // printf("Sum %f \n", e);
            // CHECK_CUDA_ERROR(hipMemPrefetchAsync(vp, bytes, deviceId));
        // }
        // n_count ++;
        // n++;



        // e = 0.0;
        // double w = 0.0;

        // for (int iy = 1; iy < (NYL+1); iy++) //! changed here
        // {
        //     for( int ix = 1; ix < (NXL+1); ix++ )
        //     {
        //         double d;

        //         vp[iy*(NXL+2)+ix] = -0.25 * (f[iy*(NXL+2)+ix] - (v[(NXL+2)*iy+ix+1] + v[(NXL+2)*iy+ix-1] + v[(NXL+2)*(iy+1) + ix  ] + v[(NXL+2)*(iy-1) + ix  ]));

        //         d = fabs(vp[iy*(NXL+2)+ix] - v[iy*(NXL+2)+ix]);
        //         e = (d > e) ? d : e;
        //         w += fabs(vp[(NXL+2)*iy+ix]);
        //     }
        // }
        // TODO can also be done non-blocking but yields worse performance??
        // send e
        // MPI_Request requests[2];
        // MPI_Iallreduce(MPI_IN_PLACE, &e, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD,&requests[0]);
        // Update v and compute error as well as error weight factor


        double* v_tmp = v;
        v = vp;
        vp = v_tmp;   

        if(n_count >= N_SKIP){
            calcDs<<<numberOfBlocks,threadsPerBlock>>>(v,vp,ds,NXL,NYL);
            e = thrust::transform_reduce(&ds_thrust[0], &ds_thrust[NXL * NYL],absolute_value<double>(), (double) 0.0, thrust::maximum<double>());
            double w = thrust::transform_reduce(&v_thrust[0], &v_thrust[NXL * NYL],absolute_value<double>(), (double) 0.0, thrust::plus<double>());
            
            // CHECK_CUDA_ERROR(hipDeviceSynchronize());         
            MPI_Allreduce(MPI_IN_PLACE, &w, 1, MPI_DOUBLE, MPI_SUM,
                MPI_COMM_WORLD);
            MPI_Allreduce(MPI_IN_PLACE, &e, 1, MPI_DOUBLE, MPI_MAX,
                    MPI_COMM_WORLD);
            // MPI_Iallreduce(MPI_IN_PLACE, &w, 1, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD,&requests[1]);

            // MPI_Waitall(2, requests, MPI_STATUSES_IGNORE);



            w /= (nx * ny);
            e /= w;
            n_count = 0;
        }
        n_count ++;
        n++;
            int counts[4] = {1, 1, 1, 1};

            /*
                share the perimeter of the inner matrix with neighbors
                    We can avoid to share the corners because we don't need them for the computation of the inner matrix
                _               _
                | - -  - - - -  - | 
                | - tl t t t tr - |
                | - l  - - - r  - |
                | - l  - - - r  - |
                | - bl b b b br - |
                |_- -  - - - -  -_|

                indices:
                    0: center-left neighbor communication
                    1: center-right neighbor communication
                    2: center-top neighbor communication
                    3: center-bottom neighbor communication
            */
            MPI_Datatype bufferTypes[4];
            bufferTypes[0] = vectorTypeVertical;     // left
            bufferTypes[1] = vectorTypeVertical;     // right
            bufferTypes[2] = contiguousTypeHorizontal; // top
            bufferTypes[3] = contiguousTypeHorizontal; // bottom
            MPI_Aint sdispls[4];
            /*
                put in sdispls where the boundaries that you want to share of the center matrix start


                |-----------------| = NXL + 2
                |-----------| = NXL
                _               _
                | - -  - - - -  - | 
                | - tl t t t tr - |
                | - l  - - - r  - |
                | - l  - - - r  - |
                | - bl b b b br - |
                |_- -  - - - -  -_|

            */
            //For example the left boundary vector of the inner matrix will be sent to the left neighbor (index 0) and it starts at tl
            sdispls[0] = ((NXL + 2) + 1) * sizeof(double);
            sdispls[1] = ((NXL + 2) + NXL) * sizeof(double);
            sdispls[2] = ((NXL + 2) + 1) * sizeof(double);
            sdispls[3] = ((NYL) * (NXL + 2) + 1) * sizeof(double);
            MPI_Aint rdispls[4];
            /*
                put in rdispls[i] the index you should start writing the boundary that you receive from communication i

                I've drawn the neighbors around the receiving matrix 
                    The matrix in the middle also share its perimeter but here we focus on the receiving part
                
                    | -  bl b b b br -  |
                    |_-  -  - - - -  - _|
                    _   _                 _    _
                -  - | | -  bl b b b br -  |  | - - 
                tr - | | tr -  - - - -  tl |  | - tl
                r  - | | r  -  - - - -  l  |  | - l 
                r  - | | r  -  - - - -  l  |  | - l 
                br - | | br -  - - - -  bl |  | - bl
                -  -_| |_-  tl t t t tr - _|  |_- - 
                        _                 _ 
                    | -  -  - - - -  -  |
                    | -  tl t t t tr -  |
            */
            //Example: from the left neighbor (i=0) it receives r vector, put it in the first column
            rdispls[0] = (NXL + 2) * sizeof(double);
            rdispls[1] = ((NXL + 2) + (NXL + 1)) * sizeof(double);
            rdispls[2] = 1 * sizeof(double);
            rdispls[3] = ((NYL + 1) * (NXL + 2) + 1) * sizeof(double);



            MPI_Request requests[8] = {MPI_REQUEST_NULL,MPI_REQUEST_NULL,MPI_REQUEST_NULL,MPI_REQUEST_NULL,MPI_REQUEST_NULL,MPI_REQUEST_NULL,MPI_REQUEST_NULL,MPI_REQUEST_NULL};
            sdispls[0] = ((NXL + 2) + 1) ;
            sdispls[1] = ((NXL + 2) + NXL) ;
            sdispls[2] = ((NXL + 2) + 1) ;
            sdispls[3] = ((NYL) * (NXL + 2) + 1);
            rdispls[0] = (NXL + 2) ;
            rdispls[1] = ((NXL + 2) + (NXL + 1)) ;
            rdispls[2] = 1 ;
            rdispls[3] = ((NYL + 1) * (NXL + 2) + 1);
            // CHECK_CUDA_ERROR(hipDeviceSynchronize()); 
            // MPI_Isend(v + sdispls[0], 1, bufferTypes[0], left, 0, comm, &(requests[0]));
            // MPI_Isend(v + sdispls[1], 1, bufferTypes[1], right, 0, comm, &(requests[1]));
            MPI_Isend(v_left, 1, contiguousTypeVertical, left, 0, comm, &(requests[0]));
            MPI_Isend(v_right, 1, contiguousTypeVertical, right, 0, comm, &(requests[1]));
            // MPI_Sendrecv_replace(v_left, 1, contiguousTypeHorizontal, left, 0,right,0, comm , MPI_STATUS_IGNORE);
            // MPI_Sendrecv_replace(v_right, 1, contiguousTypeHorizontal, right, 0,left,0, comm,MPI_STATUS_IGNORE);
            MPI_Isend(v + sdispls[2], 1, bufferTypes[2], top, 0, comm, &(requests[2]));
            MPI_Isend(v + sdispls[3], 1, bufferTypes[3], bot, 0, comm, &(requests[3]));
            // MPI_Irecv(v + rdispls[0], 1, bufferTypes[0], left, 0, comm, &(requests[4]));
            // MPI_Irecv(v + rdispls[1], 1, bufferTypes[1], right, 0, comm, &(requests[5]));
            MPI_Irecv(v_right_recv, 1, contiguousTypeVertical, right, 0, comm, &(requests[4]));
            MPI_Irecv(v_left_recv, 1, contiguousTypeVertical, left, 0, comm, &(requests[5]));
            MPI_Irecv(v + rdispls[3], 1, bufferTypes[2], top, 0, comm, &(requests[6]));
            MPI_Irecv(v + rdispls[2], 1, bufferTypes[3], bot, 0, comm, &(requests[7]));
            // # hipMemcpyPeerAsync
            MPI_Waitall(8, requests, MPI_STATUSES_IGNORE);
            // v_tmp = v_left;
            // v_left = v_left_recv;
            // v_left = v_tmp;
            // v_tmp = v_right;
            // v_right = v_right_recv;
            // v_right = v_tmp;
            jacobi_rewriteRightLeftBounds<<<numberOfBlocks,threadsPerBlock>>>(v,v_left_recv,v_right_recv,NXL+2,NYL+2);
            CHECK_LAST_CUDA_ERROR();


            // MPI_Neighbor_alltoallw(
            //     v, counts, sdispls, bufferTypes,
            //     v, counts, rdispls, bufferTypes, comm);
                


            // // Boundary conditions for w
            // if ( isLeftBorderTile ) {
            //     for (int iy = 1; iy < (NYL+1); iy++) 
            //     {
            //         w += fabs(v[(NXL+2)*iy+0]);
            //     }
            // }
            // if ( isRightBorderTile ) {
            //     for (int iy = 1; iy < (NYL+1); iy++) 
            //     {
            //         w += fabs(v[(NXL+2)*iy+((NXL+2)-1)]);
            //     }
            // }
            // if ( isTopBorderTile ) {
            //     for (int ix = 1; ix < (NXL+1); ix++) 
            //     {
            //         w += fabs(v[(NXL+2)*0+ix]);
            //     }
            // }
            // if ( isBottomBorderTile ) {
            //     for (int ix = 1; ix < (NXL+1); ix++) 
            //     {
            //         w += fabs(v[(NXL+2)*((NYL+2)-1)+ix]);
            //     }
            // }


    }
    CHECK_CUDA_ERROR(hipDeviceSynchronize()); 
    CHECK_CUDA_ERROR(hipMemcpy(v_CPU, v, bytes, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipMemcpy(vp_CPU, vp, bytes, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipMemcpy(f_CPU, f, bytes, hipMemcpyDeviceToHost));

    // return 1;
    MPI_Type_free(&vectorTypeVertical);
    MPI_Type_free(&contiguousTypeHorizontal);

    if(rank == 0)
        if (e < eps)
            printf("Converged after %d iterations (nx=%d, ny=%d, e=%.2e)\n", n, nx, ny, e);
        else
            printf("ERROR: Failed to converge\n");

    return (e < eps ? 0 : 1);
}